#include "hip/hip_runtime.h"
# include "cuda_matrix.cuh"
# include <iostream>
# include <stdlib.h>
# include <cmath>

CudaMatrix ones(size_t M, size_t N, hipblasHandle_t handle) {
  float *mat;
  hipMalloc((void**)&mat, M * N * sizeof(float));

  CudaMatrix out(handle, M, N);
  out.setMat(mat);

  out = out * 0 + 1;

  return out;
}

CudaMatrix::~CudaMatrix() {
}

CudaMatrix::CudaMatrix(hipblasHandle_t handle, size_t M, size_t N, const float* a_h) {
  hipError_t cudaStat;
  hipblasStatus_t stat;
  this->handle_ = handle;
  this->M_ = M;
  this->N_ = N;
  float *a_d_tmp;
  cudaStat = hipMalloc ((void**)&a_d_tmp, M * N * sizeof (float));
  a_d_ = std::shared_ptr<float>(a_d_tmp, hipFree);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  stat = hipblasSetMatrix(M, N, sizeof (float), a_h, M, a_d_.get(), M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    hipFree(a_d_.get());
    hipblasDestroy(handle);
    throw std::runtime_error("data download failed");
  }
}

CudaMatrix::CudaMatrix(hipblasHandle_t handle, size_t M, size_t N) {
  hipError_t cudaStat;
  this->handle_ = handle;
  this->M_ = M;
  this->N_ = N;
  float *a_d_tmp;
  cudaStat = hipMalloc((void**)&a_d_tmp, M * N * sizeof (float));
  a_d_ = std::shared_ptr<float>(a_d_tmp, hipFree);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");
}

CudaMatrix::CudaMatrix(const CudaMatrix& m) {
  hipError_t cudaStat;
  this->handle_ = m.handle_;
  this->M_ = m.M_;
  this->N_ = m.N_;
  float *a_d_tmp;
  cudaStat = hipMalloc((void**)&a_d_tmp, m.M_ * m.N_ * sizeof (float));
  this->a_d_ = std::shared_ptr<float>(a_d_tmp, hipFree);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  cudaStat = hipMemcpy(this->a_d_.get(), m.a_d_.get(), m.M_ * m.N_ * sizeof (float), hipMemcpyDeviceToDevice);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device Memcpy failed");
}

CudaMatrix& CudaMatrix::operator*(const CudaMatrix& m) const {
  CudaMatrix* c = new CudaMatrix(handle_, M_, m.N_);
  float alpha = 1.;
  float beta = 0.;
  hipblasStatus_t stat = hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, M_, m.N_, N_, &alpha, a_d_.get(), M_, m.a_d_.get(), m.N_, &beta, c->a_d_.get(), M_);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("Matrix dot product failed");
  return *c;
}

CudaMatrix& CudaMatrix::operator=(const CudaMatrix& m) {
  hipError_t cudaStat;
  this->handle_ = m.handle_;
  this->M_ = m.M_;
  this->N_ = m.N_;
  float *a_d_tmp;
  cudaStat = hipMalloc((void**)&a_d_tmp, M_ * N_ * sizeof (float));
  this->a_d_ = std::shared_ptr<float>(a_d_tmp, hipFree);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  cudaStat = hipMemcpy(this->a_d_.get(), m.a_d_.get(), m.M_ * m.N_ * sizeof (float), hipMemcpyDeviceToDevice);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device Memcpy failed");

  return *this;
}

CudaMatrix& CudaMatrix::operator*(float x) const {
  CudaMatrix *c = new CudaMatrix(handle_, M_, N_);
  hipblasStatus_t stat = hipblasSscal(handle_, c->M_ * c->N_, &x, c->a_d_.get(), 1);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("Matrix multiplication with scalar failed");
  return *c;
}

CudaMatrix& CudaMatrix::operator%(const CudaMatrix& m) const {
  CudaMatrix *c = new CudaMatrix(handle_, M_, m.N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecMulKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), c->a_d_.get(), M_ * N_);
  return *c;
}

CudaMatrix& CudaMatrix::operator+(const CudaMatrix& m) const {
  CudaMatrix *c = new CudaMatrix(handle_, 10, m.N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), c->a_d_.get(), M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *c;
}

CudaMatrix& CudaMatrix::operator-(const CudaMatrix& m) const {
  CudaMatrix* c = new CudaMatrix(handle_, M_, m.N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecSubKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), c->a_d_.get(), M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *c;
}

CudaMatrix& CudaMatrix::operator+(float m) const {
  CudaMatrix* c = new CudaMatrix(handle_, M_, N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  scalarAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m, c->a_d_.get(), M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *c;
}

CudaMatrix& CudaMatrix::operator-(float m) const {
  CudaMatrix* c = new CudaMatrix(handle_, M_, N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  scalarAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), -m, c->a_d_.get(), M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *c;
}

CudaMatrix& CudaMatrix::t() const {
  CudaMatrix* c = new CudaMatrix(handle_, N_, M_);
  float alpha = 1.;
  float beta = 0.;
  hipblasStatus_t stat = hipblasSgeam(handle_, HIPBLAS_OP_T, HIPBLAS_OP_T, M_, N_, &alpha, a_d_.get(), N_, nullptr, &beta, M_, c->a_d_.get(), N_);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("Matrix transposition failed");
  return *c;
}

CudaMatrix& CudaMatrix::transform(float (*f)(float)) {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  matTransformKernel<<<DimGrid,DimBlock>>>(a_d_.get(), f, this->M_ * this->N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *this;
}

CudaMatrix& CudaMatrix::relu() {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  matRelu<<<DimGrid,DimBlock>>>(a_d_.get(), this->M_ * this->N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *this;
}

CudaMatrix& CudaMatrix::d_relu() {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  matDRelu<<<DimGrid,DimBlock>>>(a_d_.get(), this->M_ * this->N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *this;
}

CudaMatrix& CudaMatrix::reshape(size_t M, size_t N) {
  if (M_ * N_ != M * N)
    throw std::runtime_error("Bad Reshape");
  CudaMatrix *out = new CudaMatrix(*this);
  out->M_ = M;
  out->N_ = N;
  return *out;
}

void CudaMatrix::randomize() {
  hiprandState_t* states;
  hipError_t cudaStat = hipMalloc((void**) &states, M_ * N_ * sizeof (hiprandState_t));
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  init<<<M_ * N_, 1>>>(time(0), states);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  randomizeKernel<<<DimGrid,DimBlock>>>(states, a_d_.get(), M_ * N_);
}

CudaMatrix& CudaMatrix::rows(size_t start, size_t end) const {
  CudaMatrix* c = new CudaMatrix(handle_, start - end, N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  rowGetter<<<DimGrid,DimBlock>>>(a_d_.get(), c->a_d_.get(), start, end, N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return *c;
}

CudaMatrix& CudaMatrix::rows(std::vector<size_t>& indices) const {
  CudaMatrix* c = new CudaMatrix(handle_, indices.size(), N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  for (size_t i = 0; i < indices.size(); ++i)
    rowGetter<<<DimGrid,DimBlock>>>(a_d_.get(), c->a_d_.get() + i * N_, indices[i], indices[i] + 1, N_);
  return *c;
}

float CudaMatrix::accu() const {
  return thrust::reduce(a_d_.get(), a_d_.get() + M_ * N_);
}

void CudaMatrix::addBias() {
  float* newi;
  hipError_t cudaStat = hipMalloc((void**) &newi, this->M_ * (this->N_ + 1) * sizeof(float));
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  auto tmp = a_d_.get();
  this->a_d_ = std::shared_ptr<float>(newi, hipFree);
  this->N_++;

  *this = *this * 0 + 1;

  cudaStat = hipMemcpy((void*)newi, (void*)tmp, this->M_ * (this->N_ - 1) * sizeof(float), hipMemcpyDeviceToDevice);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Memcpy failed");

  hipFree(tmp);
}

void CudaMatrix::print() const {
  float* tmp = (float*)malloc(M_ * N_ * sizeof(float));
  hipblasGetMatrix(M_, N_, sizeof(float), a_d_.get(), M_, (void *)tmp, M_);
  for (size_t i = 0; i < M_; ++i) {
    for (size_t j = 0; j < N_; ++j) {
      std::cout << tmp[i * N_ +j] << " ";
    }
    std::cout << "\n";
  }
}
