#include "cuda_matrix.cuh"

CudaMatrix::~CudaMatrix() {
  hipFree(a_d_);
}

CudaMatrix::CudaMatrix(hipblasHandle_t handle, size_t M, size_t N, const float* a_h) {
  hipError_t cudaStat;
  hipblasStatus_t stat;
  this->handle_ = handle;
  this->M_ = M;
  this->N_ = N;
  cudaStat = hipMalloc ((void**)&a_d_, M * N * sizeof (float));
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  stat = hipblasSetMatrix (M, N, sizeof (float), a_h, M, a_d_, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    hipFree(a_d_);
    hipblasDestroy(handle);
    throw std::runtime_error("data download failed");
  }
}

CudaMatrix::CudaMatrix(hipblasHandle_t handle, size_t M, size_t N) {
  hipError_t cudaStat;
  this->handle_ = handle;
  this->M_ = M;
  this->N_ = N;
  cudaStat = hipMalloc ((void**)&a_d_, M * N * sizeof (float));
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");
}

CudaMatrix::CudaMatrix(const CudaMatrix& m) {
  hipError_t cudaStat;
  this->handle_ = m.handle_;
  this->M_ = m.M_;
  this->N_ = m.N_;
  cudaStat = hipMalloc ((void**)&a_d_, m.M_ * m.N_ * sizeof (float));
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device memory allocation failed");

  cudaStat = hipMemcpy(this->a_d_, m.a_d_, m.M_ * m.N_ * sizeof (float), hipMemcpyDeviceToDevice);
  if (cudaStat != hipSuccess)
    throw std::runtime_error("Device Memcpy failed");
}

CudaMatrix CudaMatrix::operator*(const CudaMatrix& m) {
  CudaMatrix c = CudaMatrix(handle_, M_, m.N_);
  float alpha = 1.;
  float beta = 0.;
  hipblasStatus_t stat = hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, M_, m.N_, N_, &alpha, a_d_, M_, m.a_d_, m.N_, &beta, c.a_d_, M_);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("Matrix dot product failed");
  return c;
}

CudaMatrix CudaMatrix::operator*(float x) {
  CudaMatrix c = CudaMatrix(handle_, M_, N_);
  hipblasStatus_t stat = hipblasSscal(handle_, c.M_ * c.N_, &x, c.a_d_, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("Matrix multiplication with scalar failed");
  return c;
}

CudaMatrix CudaMatrix::operator%(const CudaMatrix& m) {
  CudaMatrix c = CudaMatrix(handle_, M_, m.N_);
  dim3 DimGrid((M_ * N_ - 1)/256 + 1, 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecMulKernel<<<DimGrid,DimBlock>>>(a_d_, m.a_d_, c.a_d_, M_ * N_);
  return c;
}

CudaMatrix CudaMatrix::operator+(const CudaMatrix& m) {
  CudaMatrix c = CudaMatrix(handle_, M_, m.N_);
  dim3 DimGrid((M_ * N_ - 1)/256 + 1, 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecAddKernel<<<DimGrid,DimBlock>>>(a_d_, m.a_d_, c.a_d_, M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return c;
}

CudaMatrix CudaMatrix::operator-(const CudaMatrix& m) {
  CudaMatrix c = CudaMatrix(handle_, M_, m.N_);
  dim3 DimGrid((M_ * N_ - 1)/256 + 1, 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecSubKernel<<<DimGrid,DimBlock>>>(a_d_, m.a_d_, c.a_d_, M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return c;
}

CudaMatrix CudaMatrix::operator+(float m) {
  CudaMatrix c = CudaMatrix(handle_, M_, N_);
  dim3 DimGrid((M_ * N_ - 1)/256 + 1, 1, 1);
  dim3 DimBlock(256, 1, 1);
  scalarAddKernel<<<DimGrid,DimBlock>>>(a_d_, m, c.a_d_, M_ * N_);
  hipError_t stat = hipDeviceSynchronize();
  if (stat != hipSuccess)
    throw std::runtime_error("Device synchrnization failed");
  return c;
}

CudaMatrix CudaMatrix::t() const {
  CudaMatrix c = CudaMatrix(handle_, N_, M_);
  float alpha = 1.;
  float beta = 0.;
  hipblasStatus_t stat = hipblasSgeam(handle_, HIPBLAS_OP_T, HIPBLAS_OP_T, M_, N_, &alpha, a_d_, N_, nullptr, &beta, M_, c.a_d_, N_);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("Matrix transposition failed");
  return c;
}

CudaMatrix CudaMatrix::transform(float (*f)(float)) {
  dim3 DimGrid((this->M_ * this->N_ - 1) / 256 + 1, 1, 1);
  dim3 DimBlock(256, 1, 1);
  matTransformKernel<<<DimGrid,DimBlock>>>(a_d_, f, this->M_ * this->N_);
  hipDeviceSynchronize();
  return *this;
}

CudaMatrix CudaMatrix::reshape(size_t M, size_t N) {
  if (M_ * N_ != M * N)
    throw std::runtime_error("Bad Reshape");
  CudaMatrix out(*this);
  out.M_ = M;
  out.N_ = N;
  return out;
}
