#include "hip/hip_runtime.h"
# include "cuda_matrix.cuh"
# include "cuda_error.hh"

# include <iostream>
# include <stdlib.h>
# include <cmath>

void sync_device() {
  hipError_t stat_ = hipDeviceSynchronize();
  if (stat_ != hipSuccess)
    throw std::runtime_error("Device synchronization failed");
}

CudaMatrix ones(size_t M, size_t N, hipblasHandle_t handle) {
  auto out = CudaMatrix(handle, M, N);
  hipMemset((void**)out.getMat().get(), 0, M * N * sizeof (float));
  out += 1.;

  return out;
}

CudaMatrix::~CudaMatrix() { }

void CudaMatrix::alloc() {
  if (half_) {
    half *a_d_tmp;
    CudaSafeCall(hipMalloc((void**)&a_d_tmp, M_ * N_ * sizeof (half)));
    a_d_ = std::shared_ptr<half>(a_d_tmp, hipFree);
  } else {
    float *a_d_tmp;
    CudaSafeCall(hipMalloc((void**)&a_d_tmp, M_ * N_ * sizeof (float)));
    f_d_ = std::shared_ptr<float>(a_d_tmp, hipFree);
  }
}

CudaMatrix::CudaMatrix(hipblasHandle_t handle, size_t M, size_t N, const float* a_h, bool half = false) {
  this->handle_ = handle;
  this->M_ = M;
  this->N_ = N;
  this->half_ = half;
  this->alloc();

  if (half_) {
    float *a_d_tmp;
    CudaSafeCall(hipMalloc((void**)&a_d_tmp, M_ * N_ * sizeof (float)));
    CublasSafeCall(hipblasSetMatrix(M, N, sizeof (float), a_h, M, a_d_tmp, M));

    dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
    dim3 DimBlock(256, 1, 1);
    f2h<<<DimGrid,DimBlock>>>(a_d_tmp, a_d_.get(), M_ * N_);
    CudaSafeCall(hipFree(a_d_tmp));
    sync_device();
  }
  else
    CublasSafeCall(hipblasSetMatrix(M, N, sizeof (float), a_h, M, a_d_.get(), M));
}

CudaMatrix::CudaMatrix(hipblasHandle_t handle, size_t M, size_t N, bool half = false) {
  this->handle_ = handle;
  this->M_ = M;
  this->N_ = N;
  this->half_ = half;
  this->alloc();
}

CudaMatrix::CudaMatrix(const CudaMatrix& m) {
  this->handle_ = m.handle_;
  this->M_ = m.M_;
  this->N_ = m.N_;
  this->half_ = m.half_;
  this->alloc();

  if (half_)
    CudaSafeCall(hipMemcpy(this->a_d_.get(), m.a_d_.get(), m.M_ * m.N_ * sizeof (half), hipMemcpyDeviceToDevice));
  else
    CudaSafeCall(hipMemcpy(this->f_d_.get(), m.f_d_.get(), m.M_ * m.N_ * sizeof (float), hipMemcpyDeviceToDevice));
}

// WORK
CudaMatrix CudaMatrix::operator*(const CudaMatrix& m) const {
  float alpha = 1.;
  float beta = 0.;
  auto c = CudaMatrix(handle_, M_, m.N_);

  CublasSafeCall(hipblasHgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, m.N_, M_, N_, &alpha, m.a_d_.get(), m.N_, a_d_.get(), N_, &beta, c.a_d_.get(), m.N_));

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::mult_buff(const CudaMatrix& m, CudaMatrix& o) const {
  float alpha = 1.;
  float beta = 0.;

  o.M_ = this->M_;
  o.N_ = m.N_;

  CublasSafeCall(hipblasHgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, m.N_, M_, N_, &alpha, m.a_d_.get(), m.N_, a_d_.get(), N_, &beta, o.a_d_.get(), m.N_));

  sync_device();

  return o;
}

// WORK
CudaMatrix CudaMatrix::dot(const CudaMatrix& m, float alpha) const {
  float beta = 0.;
  auto c = CudaMatrix(handle_, M_, m.N_);

  CublasSafeCall(hipblasHgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, m.N_, M_, N_, &alpha, m.a_d_.get(), m.N_, a_d_.get(), N_, &beta, c.a_d_.get(), m.N_));

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::operator=(const CudaMatrix& m) {

  this->M_ = m.M_;
  this->N_ = m.N_;
  this->a_d_ = m.a_d_;
  this->f_d_ = m.f_d_;
  this->half_ = m.half_;

  return *this;
}

// WORK
CudaMatrix CudaMatrix::operator*(float x) const {
  auto c = CudaMatrix(*this);
  CublasSafeCall(hipblasSscal(handle_, c.M_ * c.N_, &x, c.a_d_.get(), 1));

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::operator%(const CudaMatrix& m) const {
  if (this->shape() != m.shape()) {
    std::cout << "% failed\n";
    this->print_shape("this\t");
    m.print_shape("m\t");
    exit(-1);
  }

  auto c = CudaMatrix(handle_, M_, m.N_);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecMulKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), c.a_d_.get(), M_ * N_);

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::operator%=(const CudaMatrix& m) {
  if (this->shape() != m.shape()) {
    std::cout << "% failed\n";
    this->print_shape("this\t");
    m.print_shape("m\t");
    exit(-1);
  }

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecMulKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), a_d_.get(), M_ * N_);

  sync_device();

  return *this;
}

// WORK
CudaMatrix CudaMatrix::operator+(const CudaMatrix& m) const {
  if (this->shape() != m.shape()) {
    std::cout << "+ failed\n";
    this->print_shape("this\t");
    m.print_shape("m\t");
  }

  CudaMatrix c = CudaMatrix(handle_, m.M_, m.N_);

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), c.a_d_.get(), M_ * N_);

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::operator+=(const CudaMatrix& m) {
  if (this->shape() != m.shape()) {
    std::cout << "+ failed\n";
    this->print_shape("this\t");
    m.print_shape("m\t");
  }

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  vecAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), a_d_.get(), M_ * N_);

  sync_device();

  return *this;
}

// WORK
CudaMatrix CudaMatrix::operator-(const CudaMatrix& m) const {
  dim3 DimGrid(std::ceil((N_ * M_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);

  auto c = CudaMatrix(handle_, M_, N_);
  vecSubKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), c.a_d_.get(), M_ * N_);
  sync_device();
  return c;
}

// WORK
CudaMatrix CudaMatrix::operator-=(const CudaMatrix& m) {
  dim3 DimGrid(std::ceil((N_ * M_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);

  vecSubKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m.a_d_.get(), a_d_.get(), M_ * N_);
  sync_device();
  return *this;
}

// WORK
CudaMatrix CudaMatrix::operator+(float m) const {
  auto c = CudaMatrix(handle_, M_, N_);

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  scalarAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m, c.a_d_.get(), M_ * N_);

  sync_device();

  return c;
}

CudaMatrix CudaMatrix::operator+=(float m) {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  scalarAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), m, a_d_.get(), M_ * N_);

  sync_device();

  return *this;
}

// WORK
CudaMatrix CudaMatrix::operator-(float m) const {
  auto c = CudaMatrix(handle_, M_, N_);

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  scalarAddKernel<<<DimGrid,DimBlock>>>(a_d_.get(), -m, c.a_d_.get(), M_ * N_);

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::t() const {
  auto c = CudaMatrix(handle_, N_, M_);

  float alpha = 1.;
  float beta = 0.;

  CublasSafeCall(cublasHgeam(handle_, HIPBLAS_OP_T, HIPBLAS_OP_T, M_, N_, &alpha, this->a_d_.get(), N_, &beta, this->a_d_.get(), N_, c.a_d_.get(), M_));

  sync_device();
  return c;
}

CudaMatrix CudaMatrix::transform(float (*f)(float)) {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  matTransformKernel<<<DimGrid,DimBlock>>>(a_d_.get(), f, this->M_ * this->N_);

  sync_device();

  return *this;
}

// WORK
CudaMatrix CudaMatrix::relu() {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  matTanh<<<DimGrid,DimBlock>>>(a_d_.get(), this->M_ * this->N_);

  sync_device();

  return *this;
}

// WORK
CudaMatrix CudaMatrix::d_relu() {
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  matDTanh<<<DimGrid,DimBlock>>>(a_d_.get(), this->M_ * this->N_);

  sync_device();

  return *this;
}

// WORK
CudaMatrix CudaMatrix::reshape(size_t M, size_t N) {
  if (M_ * N_ != M * N)
    throw std::runtime_error("Bad Reshape");

  CudaMatrix out = CudaMatrix(*this);
  out.M_ = M;
  out.N_ = N;

  return out;
}

// WORK
void CudaMatrix::randomize() {
  hiprandState_t* states;
  CudaSafeCall(hipMalloc((void**) &states, M_ * N_ * sizeof (hiprandState_t)));

  init<<<M_ * N_, 1>>>(time(0), states);

  sync_device();

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  randomizeKernel<<<DimGrid,DimBlock>>>(states, a_d_.get(), M_ * N_);

  sync_device();
}

// WORK
CudaMatrix CudaMatrix::rows(size_t start, size_t end) const {
  auto c = CudaMatrix(handle_, end - start, N_);

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  rowGetter<<<DimGrid,DimBlock>>>(a_d_.get(), c.a_d_.get(), start, end, N_);

  sync_device();

  return c;
}

// WORK
CudaMatrix CudaMatrix::rows(std::vector<size_t>& indices) const {
  auto c = CudaMatrix(handle_, indices.size(), N_);

  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  for (size_t i = 0; i < indices.size(); ++i)
    rowGetter<<<DimGrid,DimBlock>>>(a_d_.get(), c.a_d_.get() + i * N_, indices[i], indices[i] + 1, N_);

  sync_device();

  return c;
}

// WORK
float CudaMatrix::accu() const {
  float *a_d_tmp;
  CudaSafeCall(hipMalloc((void**)&a_d_tmp, M_ * N_ * sizeof (float)));
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  h2f<<<DimGrid,DimBlock>>>(a_d_get, a_d_tmp, M_ * N_);
  CudaSafeCall(hipFree(a_d_tmp));
  sync_device();
  return thrust::reduce(thrust::device, a_d_.get(), a_d_.get() + M_ * N_);
}

// WORK
CudaMatrix CudaMatrix::addBias() {
  auto out = ones(this->M_, this->N_ + 1, handle_);

  for (size_t i = 0; i < this->M_; ++i)
    CudaSafeCall(hipMemcpy(out.a_d_.get() + i * (this->N_ + 1), this->a_d_.get() + i * N_, N_ * sizeof (float), hipMemcpyDeviceToDevice));

  sync_device();

  return out;
}

void CudaMatrix::print() const {
  float* tmp = (float*)malloc(M_ * N_ * sizeof (float));
  hipblasGetMatrix(M_, N_, sizeof (float), a_d_.get(), M_, (void *)tmp, M_);
  for (size_t i = 0; i < M_; ++i) {
    for (size_t j = 0; j < N_; ++j) {
      std::cout << tmp[i * N_ +j] << " ";
    }
    std::cout << "\n";
  }
  std::cout << "\n";
  free(tmp);
}

std::pair<size_t, size_t> CudaMatrix::shape() const {
  return std::pair<size_t, size_t>(M_, N_);
}

void CudaMatrix::print_shape(std::string str) const {
  std::cout << str << this->M_ << ":" << this->N_ << std::endl;
}

CudaMatrix CudaMatrix::getHalf() const {
  if (half_) {
    std::cout << "This is a half precision matrix." << std::endl;
    exit(1);
  }
  auto ans = CudaMatrix(handle_, M_, N_, true);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  f2h<<<DimGrid,DimBlock>>>(f_d_.get(), aux.a_d_.get(), M_ * N_);
  CudaSafeCall(hipFree(a_d_tmp));
  sync_device();
}

CudaMatrix CudaMatrix::getHalf() const {
  if (!half_) {
    std::cout << "This is a sigle precision matrix." << std::endl;
    exit(1);
  }
  auto ans = CudaMatrix(handle_, M_, N_, false);
  dim3 DimGrid(std::ceil((M_ * N_) / 256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  h2f<<<DimGrid,DimBlock>>>(a_d_.get(), aux.f_d_.get(), M_ * N_);
  CudaSafeCall(hipFree(a_d_tmp));
  sync_device();
}
