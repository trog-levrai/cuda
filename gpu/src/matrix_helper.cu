#include "hip/hip_runtime.h"
__global__ void vecMulKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i<n) C[i] = A[i] * B[i];
}

__global__ void matTransformKernel(float* A, std::function<float (float)> f, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i<n) A[i] = f(A[i]);
}
