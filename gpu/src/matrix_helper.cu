#include "hip/hip_runtime.h"
#include "matrix_helper.cuh"

__global__ void vecMulKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] * B[i];
}

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] + B[i];
}

__global__ void vecSubKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] - B[i];
}

__global__ void scalarAddKernel(float* A, float s, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] + s;
}

template<typename Lambda>
__global__ void matTransformKernel(float* A, float (*f)(float), int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) A[i] = f(A[i]);
}
