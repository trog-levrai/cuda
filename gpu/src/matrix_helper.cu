#include "hip/hip_runtime.h"
#include "matrix_helper.cuh"

__global__ void vecMulKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] * B[i];
}

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] + B[i];
}

__global__ void vecSubKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] - B[i];
}

__global__ void scalarAddKernel(float* A, float s, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) C[i] = A[i] + s;
}

__global__ void matTransformKernel(float* A, float (*f)(float), int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) A[i] = f(A[i]);
}

__global__ void init(unsigned int seed, hiprandState_t* states) {
  hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void randomizeKernel(hiprandState_t* states, float* a, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i<n) a[i] = hiprand_uniform(&states[i]);
}
